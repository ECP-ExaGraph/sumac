
using namespace std;

#include <cstdlib>
#include <iostream>
#include <assert.h>
#include <cmath>
#include <cstring>
#include <chrono>
#include <random>
#include <omp.h>

#include "io.h"
#include "graph.h"
#include "match.h"

int VertsPerWarp = 8;

int seed = 0;

int main(int argc, char** argv)
{
  setbuf(stdout, NULL);
  srand(time(0));

  if (argc < 1)
  {
    printf("To run: %s [graphFile]\n\n",
      argv[0]);
    exit(0);
  }

  char* graphFile = argv[1];
  graph* g_host = create_graph(graphFile);

  graph* g = NULL;
  assert(hipMallocManaged(&g, sizeof(graph)) == hipSuccess);
  hipMemcpy(&g->num_verts, &g_host->num_verts, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(&g->num_edges, &g_host->num_edges, sizeof(long), hipMemcpyHostToDevice);
  hipMemcpy(&g->max_degree, &g_host->max_degree, sizeof(long), hipMemcpyHostToDevice);

  long num_verts = g_host->num_verts;
  long num_edges = g_host->num_edges;
  assert(hipMallocManaged(&g->out_adjlist, num_edges*sizeof(int)) == hipSuccess);
  assert(hipMallocManaged(&g->out_offsets, (num_verts + 1)*sizeof(long)) == hipSuccess);
  hipMemcpy(g->out_adjlist, g_host->out_adjlist, num_edges*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(g->out_offsets, g_host->out_offsets, (num_verts + 1)*sizeof(long), hipMemcpyHostToDevice);
  

  printf("Graph created: %s\n",graphFile);
  volatile double** weights;
  hipMallocManaged(&weights,g->num_verts*sizeof(double));

  
  #pragma omp parallel for
  for(int u=0;u<g->num_verts;u++){
    int deg = out_degree(g,u);
    hipMallocManaged(&weights[u],deg*sizeof(double));
    for(int v=0;v<deg;v++){
      weights[u][v] = rand() % 1000;
    }
  }


  int* d_mate;
  int* h_mate = new int[g->num_verts];
  volatile int* pointers;
  int* h_verts = new int[g->num_verts];
  int* d_verts;
  hipMallocManaged(&pointers,g->num_verts*sizeof(int));

  for(long u=0;u<g->num_verts;u++){
    h_verts[u] = u;
    h_mate[u]=-1;
    pointers[u] = -1;
  }
  hipMalloc(&d_mate,g->num_verts*sizeof(int));
  hipMemcpy(d_mate,h_mate, g->num_verts*sizeof(int), hipMemcpyHostToDevice);
  hipMalloc(&d_verts, g->num_verts * sizeof(int));
  hipMemcpy(d_verts, h_verts, g->num_verts * sizeof(int), hipMemcpyHostToDevice);


  int device = -1;
  hipGetDevice(&device);
  hipMemPrefetchAsync(g->out_adjlist, num_edges*sizeof(int), device, NULL);
  hipMemPrefetchAsync(g->out_offsets, (num_verts + 1)*sizeof(long), device, NULL);

  printf("Matching data structs initialized\n");
  int block_size = 128;
  //int num_blocks = (g->num_verts + block_size - 1) / block_size;
  int num_blocks = g->num_verts / (VertsPerWarp * (block_size/32)) + 1;
  printf("NumBlocks:%ld\n",num_blocks);
  printf("Starting Matching\n");
  
  double elt = omp_get_wtime();
  hipDeviceSynchronize();
  Pointer_Chase_GPU<<<num_blocks,block_size,(VertsPerWarp*(block_size/32)+1)*sizeof(int)>>>(g,d_verts,weights,d_mate,pointers,VertsPerWarp);
  hipDeviceSynchronize();
  hipError_t error = hipGetLastError();
  if(error!=hipSuccess)
  {
    fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
    exit(-1);
  }
  printf("Finished Matching, Time: %9.6f(s)\n",omp_get_wtime()-elt);

  clear_graph(g_host);
  hipFree(g_host->out_adjlist);
  hipFree(g_host->out_offsets);
  hipFree(d_mate);
  hipFree(weights);
  return 0;
}
