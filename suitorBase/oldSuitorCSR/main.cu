
using namespace std;

#include <cstdlib>
#include <iostream>
#include <assert.h>
#include <cmath>
#include <cstring>
#include <chrono>
#include <random>
#include <omp.h>

#include "io.h"
#include "graph.h"
#include "metric.h"
#include "match.h"

int VertsPerWarp = 8;

int seed = 0;

int main(int argc, char** argv)
{
  setbuf(stdout, NULL);
  srand(time(0));

  if (argc < 1)
  {
    printf("To run: %s [graphFile]\n\n",
      argv[0]);
    exit(0);
  }

  char* graphFile = argv[1];
  graph* g_host = create_graph(graphFile);

  graph* g = NULL;
  assert(hipMallocManaged(&g, sizeof(graph)) == hipSuccess);
  hipMemcpy(&g->num_verts, &g_host->num_verts, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(&g->num_edges, &g_host->num_edges, sizeof(long), hipMemcpyHostToDevice);
  hipMemcpy(&g->max_degree, &g_host->max_degree, sizeof(long), hipMemcpyHostToDevice);

  long num_verts = g_host->num_verts;
  long num_edges = g_host->num_edges;
  assert(hipMallocManaged(&g->out_adjlist, num_edges*sizeof(int)) == hipSuccess);
  assert(hipMallocManaged(&g->out_offsets, (num_verts + 1)*sizeof(long)) == hipSuccess);
  hipMemcpy(g->out_adjlist, g_host->out_adjlist, num_edges*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(g->out_offsets, g_host->out_offsets, (num_verts + 1)*sizeof(long), hipMemcpyHostToDevice);
  

  printf("Graph created: %s\n",graphFile);

  //double** weights = new double*[g->num_verts];
  double** weights;
  hipMallocManaged(&weights,g->num_verts*sizeof(double));
  #pragma omp parallel for
  for(long u=0;u<g->num_verts;u++){
    long deg = out_degree(g,u);
    //weights[u] = new double[deg];
    hipMallocManaged(&weights[u],deg*sizeof(double));
    for(long v=0;v<deg;v++)
      weights[u][v] = rand() % 1000;
  }
  //long* mate = new long[g->num_verts];
  int* mate;
  double* ws;
  long* h_verts = new long[g->num_verts];
  long* d_verts;
  hipMallocManaged(&mate,g->num_verts*sizeof(int));
  hipMallocManaged(&ws,g->num_verts*sizeof(double));

  for(long u=0;u<g->num_verts;u++){
    h_verts[u] = u;
    mate[u]=-1;
  }


  hipMalloc(&d_verts, g->num_verts * sizeof(long));
  hipMemcpy(d_verts, h_verts, g->num_verts * sizeof(long), hipMemcpyHostToDevice);


  int device = -1;
  hipGetDevice(&device);
  hipMemPrefetchAsync(g->out_adjlist, num_edges*sizeof(int), device, NULL);
  hipMemPrefetchAsync(g->out_offsets, (num_verts + 1)*sizeof(long), device, NULL);
  int* vlocks;
  hipMalloc(&vlocks,g->num_verts*sizeof(int));

  printf("Matching data structs initialized\n");
  int block_size = 128;
  int num_blocks = (g->num_verts + block_size - 1) / block_size;
  printf("NumBlocks:%ld\n",num_blocks);
  printf("Starting Matching\n");

  double elt = omp_get_wtime();
  GPU_Suitor_Matching<<<num_blocks,block_size,(VertsPerWarp*(block_size/32))*sizeof(int)>>>(g,weights,mate,ws,VertsPerWarp,vlocks);
  hipDeviceSynchronize();
  hipError_t error = hipGetLastError();
if(error!=hipSuccess)
{
   fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
   exit(-1);
}
  printf("Finished Matching, Time: %9.6f(s)\n",omp_get_wtime()-elt);

  clear_graph(g_host);
  hipFree(g_host->out_adjlist);
  hipFree(g_host->out_offsets);
  hipFree(mate);
  hipFree(weights);
  hipFree(vlocks);
  return 0;
}
